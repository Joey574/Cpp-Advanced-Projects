#include "hip/hip_runtime.h"
#include "cuda_implementations.cuh"

__global__ void cu_dot_prod(float* a, float* b, float* c, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int row = i / n;
	int col = i % n;

	if (row < n) {
		float sum = 0.0f;

		for (int k = 0; k < n; k++) {
			sum = __fmaf_ieee_rn(a[row * n + k], b[k * n + col], sum);
			//sum += a[i * n + k] * b[k * n + j];
		}

		c[row * n + col] = sum;
	}
}

void call_kernal(float* a, float* b, float* c, int n, dim3 dim_grid, dim3 dim_block) {
	cu_dot_prod <<< dim_grid, dim_block >>> (a, b, c, n);
}