#include "hip/hip_runtime.h"
#include "cuda_implementations.cuh"

__global__ void cu_dot_prod(float* a, float* b, float* c, int n) {
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n && j < n) {
		float sum = 0.0f;

		for (int k = 0; k < n; k++) {
			sum += a[i * n + k] * b[k * n + j];
		}

		c[i * n + j] = sum;
	}
}

void call_kernal(float* a, float* b, float* c, int n, dim3 dim_grid, dim3 dim_block) {
	cu_dot_prod <<< dim_grid, dim_block >>> (a, b, c, n);
}